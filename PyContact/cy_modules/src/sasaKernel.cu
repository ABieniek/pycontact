
#include <hip/hip_runtime.h>
#include <stdio.h>

void __global__ SasaKernel(int natoms, float pairdist, const int npts, float srad, float3* pos,
    float* radius, float3* points, float* sasa)
{
    const int thread = blockDim.x * blockIdx.x + threadIdx.x;
    if (thread < natoms) {
        float3 center = pos[thread];
        float currentRadius = radius[thread];

        // Testing overlap for each point
        float3 scaledPoint;
        int remainingPoints = npts;

        for (int i = 0; i < npts; i++) {
            scaledPoint.x = points[i].x * (currentRadius + srad) + center.x;
            scaledPoint.y = points[i].y * (currentRadius + srad) + center.y;
            scaledPoint.z = points[i].z * (currentRadius + srad) + center.z;

            bool overlap = false;

            for (int atomId = 0; atomId < natoms; atomId++) {
                if (atomId != thread) {
                    float neighbourRad2 = radius[atomId] + srad;
                    neighbourRad2 *= neighbourRad2;
                    float3 neighbourCenter = pos[atomId];
                    float3 dr;
                    dr.x = scaledPoint.x - neighbourCenter.x;
                    dr.y = scaledPoint.y - neighbourCenter.y;
                    dr.z = scaledPoint.z - neighbourCenter.z;

                    // if (thread == 99 && atomId == 1000) {
                    //     printf("%f %f %f\n", dr.x, dr.y, dr.z);
                    // }

                    if (dr.x*dr.x + dr.y*dr.y + dr.z*dr.z <= neighbourRad2) {
                        overlap = true;
                        break;
                    }
                }
            }
            if (overlap) {
                remainingPoints--;
            }
        }
        sasa[thread] = 12.5663706144 * powf(currentRadius+srad, 2) * (float)(remainingPoints) / npts;
    }
}
